#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include "reflection.cu"

////////////////////////////////////////////////////////////////////////

static int Width;
static int Height;

static bool IsLoaded = false;

static hipEvent_t start;
static hipEvent_t stop;

////////////////////////////////////////////////////////////////////////

static Reflection<float> Buffer;
static Reflection<int> Frame;

static Reflection<float> MaxValueBuffer;
static Reflection<float> MinValueBuffer;
static Reflection<float> SumBuffer;

static Reflection<float> MaxValue;
static Reflection<float> MinValue;

static Reflection<float> Sum;

////////////////////////////////////////////////////////////////////////

__inline__ __device__ int Color(const int R, const int G, const int B)
{
	return (-16777216) | (R << 16) | (G << 8) | B;
}

////////////////////////////////////////////////////////////////////////

__inline__ __host__ __device__ unsigned int GetBufferIndex(const unsigned int dim, int x, int y, const unsigned int Width, const unsigned int Height)
{
    if(x < 0)
	{
		x = x % Width + Width;
	}

	if(x >= Width)
	{
		x = x % Width;
	}

    if(y < 0)
	{
		y = y % Height + Height;
	}

	if(y >= Height)
	{
		y = y % Height;
	}

	// Buffer[3][Width][Height];

	return dim*Width*Height + x*Height + y;
}

__inline__ __host__ __device__ unsigned int GetFrameIndex(const unsigned int x, const unsigned int y, const unsigned int Width, const unsigned int Height)
{
	// Frame[Width][Height];

	return x*Height + y;
}

////////////////////////////////////////////////////////////////////////

typedef int (*ColorInterpretator)(float value, float maxValue, float minValue, float WaterLevel);

namespace ColorInterpretators
{
	////////////////////////////////////////////////////////////////////////

	__device__ int DefaultColor(float value, float MaxValue, float MinValue, float WaterLevel)
	{
		if(value == 0.0f)
		{
			return Color(255, 255, 255);
		}

		if(value < 0.0f)
		{
			int intensity = (int)(255.0f * (value / MinValue));
		
			return Color(0, 0, intensity);
		}
		else
		{
			int intensity = (int)(255.0f-255.0f * (value / MaxValue));
		
			return Color(intensity, intensity, intensity);
		}
	}

	////////////////////////////////////////////////////////////////////////

	__device__ ColorInterpretator Interpretators[] = 
	{
		DefaultColor,
	};

	const unsigned int Count = 1;

	char* Titles[] = 
	{
		"DefaultColor",
	};
}

////////////////////////////////////////////////////////////////////////

__global__ void CudaSample(float* Buffer, 
						   const unsigned int Width, 
						   const unsigned int Height, 
                           const float phaseSpeed)
{
	/// <<<Width, Height>>>

    const unsigned int block = blockIdx.x;
    const unsigned int thread = threadIdx.x;

    if(block >= Width || thread >= Height)
    {
        return;
    }

	const float laplacian = Buffer[GetBufferIndex(1, block+1, thread, Width, Height)] + 
		                    Buffer[GetBufferIndex(1, block-1, thread, Width, Height)] +
		                    Buffer[GetBufferIndex(1, block, thread+1, Width, Height)] + 
		                    Buffer[GetBufferIndex(1, block, thread-1, Width, Height)] - 4.0f * 
		                    Buffer[GetBufferIndex(1, block, thread, Width, Height)];

	Buffer[GetBufferIndex(2, block, thread, Width, Height)] = 2.0f*Buffer[GetBufferIndex(1, block, thread, Width, Height)] + phaseSpeed*laplacian;
}

__global__ void PushBuffers(float* Buffer)
{
	///	<<<1, 1>>>


}

////////////////////////////////////////////////////////////////////////

__global__ void ReCountPart1(float* Buffer, 
							 float* MaxValueBuffer, 
							 float* MinValueBuffer, 
							 float* SumBuffer, 
							 const unsigned int Width, 
							 const unsigned int Height)
{
	/// <<<Width, 1>>>

	const unsigned int block = blockIdx.x;
    const unsigned int thread = threadIdx.x;

    if(block >= Width || thread > 0)
    {
        return;
    }

	////////////////////////////////////////////////////////////////////////

	float max = -FLT_MAX;
	float min = FLT_MAX;

	float sum = 0.0f;

	for(int i=0; i<Height; ++i)
	{
		const float value = Buffer[GetBufferIndex(1, block, i, Width, Height)];

		if(value < min)
		{
			min = value;
		}

		if(value > max)
		{
			max = value;
		}

		sum += value;
	}

	MinValueBuffer[block] = min;
	MaxValueBuffer[block] = max;

	SumBuffer[block] = sum;
}

__global__ void ReCountPart2(float* Buffer, 
							 float* MaxValueBuffer, 
							 float* MinValueBuffer, 
							 float* SumBuffer, 
							 float* MaxValue, 
							 float* MinValue, 
							 float* Sum, 
							 const unsigned int Width, 
							 const unsigned int Height)
{
	/// <<<1, 1>>>

	const unsigned int block = blockIdx.x;
    const unsigned int thread = threadIdx.x;

    if(block || thread)
    {
        return;
    }

	////////////////////////////////////////////////////////////////////////

	float max = -FLT_MAX;
	float min = FLT_MAX;

	float sum = 0.0f;

	for(int i=0; i<Width; ++i)
	{
		if(MaxValueBuffer[i] > max)
		{
			max = MaxValueBuffer[i];
		}

		if(MinValueBuffer[i] < min)
		{
			min = MinValueBuffer[i];
		}

		sum += SumBuffer[i];
	}

	MaxValue[0] = max;
	MinValue[0] = min;

	Sum[0] = sum;
}

////////////////////////////////////////////////////////////////////////

__global__ void CudaFrame(float* Buffer, 
						  int* Frame,
						  float* MaxValue, 
						  float* MinValue, 						  
						  const float WaterLevel,
						  const unsigned int InterpretatorIndex,
						  const unsigned int Width, 
						  const unsigned int Height)
{
	/// <<<Width, Height>>>

    const unsigned int block = blockIdx.x;
    const unsigned int thread = threadIdx.x;

    if(block >= Width || thread >= Height)
    {
        return;
    }

	const float value = Buffer[GetBufferIndex(1, block, thread, Width, Height)];

	Frame[GetFrameIndex(block, thread, Width, Height)] = ColorInterpretators::Interpretators[InterpretatorIndex](value, MaxValue[0], MinValue[0], WaterLevel);
}

void ReCount()
{
	ReCountPart1<<<Width, 1>>>(Device(Buffer), 
							   Device(MaxValueBuffer), 
							   Device(MinValueBuffer), 
							   Device(SumBuffer), 
							   Width, 
							   Height);

	ReCountPart2<<<1, 1>>>(Device(Buffer), 
						   Device(MaxValueBuffer), 
						   Device(MinValueBuffer), 
						   Device(SumBuffer), 
						   Device(MaxValue), 
						   Device(MinValue), 
						   Device(Sum), 
						   Width, 
						   Height);
}

////////////////////////////////////////////////////////////////////////

static void CudaMalloc(int Width, int Height)
{
	hipSetDevice(0);

	Buffer = Malloc<float>(3*Width*Height);

	Frame = Malloc<int>(Width*Height);

	MaxValueBuffer = Malloc<float>(Width);
	MinValueBuffer = Malloc<float>(Width);

	SumBuffer = Malloc<float>(Width);

	MaxValue = Malloc<float>(1);
	MinValue = Malloc<float>(1);

	Sum = Malloc<float>(1);
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

void CudaFree()
{
	Free(Buffer);
	Free(Frame);

	Free(MaxValueBuffer);
	Free(MinValueBuffer);

	Free(SumBuffer);

	IsLoaded = false;
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

bool CudaStart(int width, int height)
{
	if(IsLoaded)
	{
		CudaFree();
	}

	if(width < 3 || height < 3 || height > 1024)
	{
		return false;
	}

	Width = width;
	Height = height;
	
	CudaMalloc(width, height);

	IsLoaded = true;

	return IsValid(Buffer);
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

bool CudaSetState(float* buffer, int width, int height)
{
	if(buffer == nullptr || width < 3 || height < 3)
	{
		return false;
	}

	if(width != Width || height != Height)
	{
		CudaStart(width, height);
	}

	if(!IsValid(Buffer) || !IsLoaded)
	{
		return false;
	}

	const unsigned int size = 2*width*height*sizeof(float);

	memcpy(Host(Buffer), buffer, size);

	return Send(Buffer);
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

bool SetDefaultState()
{
	if(!IsValid(Buffer) || !IsLoaded)
	{
		return false;
	}

	Host(Buffer)[GetBufferIndex(0, Width >> 1, Height >> 1, Width, Height)] = 1.0f;
	Host(Buffer)[GetBufferIndex(1, Width >> 1, Height >> 1, Width, Height)] = 1.0f;

	return Send(Buffer);
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

int GetCurrentFrame(int* frame, int ColorInterpretatorIndex, float WaterLevel)
{
	if(!IsLoaded || !IsValid(Buffer) || !IsValid(Frame))
	{
		return -1;
	}

	////////////////////////////////////////////////////////////////////////

	if(ColorInterpretatorIndex > ColorInterpretators::Count)
	{
		ColorInterpretatorIndex = 0;
	}

	ColorInterpretator Selected = ColorInterpretators::Interpretators[ColorInterpretatorIndex];

	////////////////////////////////////////////////////////////////////////
	
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	////////////////////////////////////////////////////////////////////////

	ReCount();
	
	if(Height <= 1024)
	{
		CudaFrame<<<Width, Height>>>(Device(Buffer), 
									 Device(Frame),
									 Device(MaxValue), 
									 Device(MinValue), 									 
									 WaterLevel, 
									 ColorInterpretatorIndex, 
									 Width, 
									 Height);
									 
	}

	if(!Receive(Frame))
	{
		return -1;
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time = 0;

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	memcpy(frame, Host(Frame), Frame.size);

	return Frame.size;
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

int CudaCalc(float phaseSpeed)
{
	if(!IsLoaded || !IsValid(Buffer) || !IsValid(Frame))
	{
		return -1;
	}

	////////////////////////////////////////////////////////////////////////

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	////////////////////////////////////////////////////////////////////////

	if(Height <= 1024)
	{
		CudaSample<<<Width, Height>>>(Device(Buffer), Width, Height, phaseSpeed);
	}

	////////////////////////////////////////////////////////////////////////

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float time = 0;

	hipEventElapsedTime(&time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	return (int)(time+0.5f);
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

int GetColorInterpretatorCount()
{
	return ColorInterpretators::Count;
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

int GetColorInterpretatorTitle(char* str, int ColorInterpretatorIndex)
{
	if(ColorInterpretatorIndex > ColorInterpretators::Count)
	{
		return 0;
	}

	int len = strlen(ColorInterpretators::Titles[ColorInterpretatorIndex]);
	memcpy(str, ColorInterpretators::Titles[ColorInterpretatorIndex], len);	
	return len;
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport)

bool GetCudaStatus(int width, int height)
{
	bool Status = IsLoaded;

	Status = Status && IsValid(Buffer);
	Status = Status && IsValid(Frame);

	Status = Status && IsValid(MaxValueBuffer);
	Status = Status && IsValid(MinValueBuffer);
	Status = Status && IsValid(SumBuffer);

	Status = Status && IsValid(MaxValue);
	Status = Status && IsValid(MinValue);
	Status = Status && IsValid(Sum);

	Status = Status && width == Width;
	Status = Status && height == Height;

	return Status;
}

////////////////////////////////////////////////////////////////////////

extern "C" __declspec(dllexport) 

float GetSum()
{
	if(Receive(Sum))
	{
		return Host(Sum)[0];
	}

	return 0.0f;
}

////////////////////////////////////////////////////////////////////////





